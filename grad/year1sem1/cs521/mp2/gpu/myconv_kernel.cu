#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// example
#define TILE_H 8   
#define TILE_W 8   
#define TILE_C 16  

// Kernel declaration
__global__ void gemm_gpu_o4_kernel(
    const float* __restrict__ x,       // input: N x C x H x W
    const float* __restrict__ w,       // weights: C_out x C_in x KH x KW
    float* __restrict__ out,           // output: N x C x H x W
    int N, int C_in, int H, int W,
    int C_out, int KH, int KW,
    int stride, int pad,
    int out_h, int out_w
) {
    extern __shared__ float shmem[];  // shared memory for partial sums
    
    // TO DO : Tiled matrix multiplication by using shmem
}

// Function for Python binding
torch::Tensor conv_cuda(torch::Tensor x, torch::Tensor w,
                          int stride, int pad) {
    int N = x.size(0);
    int C_in = x.size(1);
    int H = x.size(2);
    int W = x.size(3);

    int C_out = w.size(0);
    int KH = w.size(2);
    int KW = w.size(3);

    // int out_h = ...
    // int out_w = ...

    auto out = torch::zeros({N, C_out, out_h, out_w}, x.options());

    dim3 block(8, 8);
    dim3 grid((out_w + block.x - 1)/block.x,
              (out_h + block.y - 1)/block.y,
              N);

    gemm_gpu_o4_kernel<<<grid, block>>>(
        x.data_ptr<float>(),
        w.data_ptr<float>(),
        out.data_ptr<float>(),
        N, C_in, H, W,
        C_out, KH, KW,
        stride, pad,
        out_h, out_w);

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("conv_cuda", &conv_cuda, "Custom Conv2D (CUDA)");
}
